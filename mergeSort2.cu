#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define THREADS_PER_BLOCK 256

// CUDA kernel to merge two sorted halves of an array
__device__ void merge(int *arr, int *temp, int l, int m, int r)
{
    int n1 = m - l + 1;
    int n2 = r - m;

    int i = 0, j = 0, k = l;
    while (i < n1 && j < n2)
    {
        if (arr[l + i] <= arr[m + 1 + j])
            temp[k++] = arr[l + i++];
        else
            temp[k++] = arr[m + 1 + j++];
    }

    while (i < n1)
        temp[k++] = arr[l + i++];

    while (j < n2)
        temp[k++] = arr[m + 1 + j++];

    for (i = l; i <= r; i++)
        arr[i] = temp[i];
}

// CUDA kernel to perform merge sort on the device
__global__ void mergeSort(int *arr, int *temp, int size)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int start = tid * size;

    for (int curr_size = 1; curr_size < size; curr_size *= 2)
    {
        for (int left_start = start; left_start < size - 1; left_start += 2 * curr_size)
        {
            int mid = left_start + curr_size - 1;
            int right_end = min(left_start + 2 * curr_size - 1, size - 1);

            if (mid < right_end)
                merge(arr, temp, left_start, mid, right_end);
        }
    }
}

int main(int argc, char *argv[])
{
    if (argc != 2)
    {
        printf("Usage: %s <array_size>\n", argv[0]);
        return 1;
    }

    int n = atoi(argv[1]);
    if (n <= 0)
    {
        printf("Invalid array size\n");
        return 1;
    }

    int *arr, *d_arr, *temp;
    arr = (int *)malloc(n * sizeof(int));
    temp = (int *)malloc(n * sizeof(int));

    // Generate random input data (for demonstration)
    if (arr == NULL || temp == NULL)
    {
        printf("Memory allocation failed\n");
        return 1;
    }

    FILE *input_file = fopen("array1.txt", "r");
    if (input_file == NULL)
    {
        printf("Error opening file: array1.txt\n");
        free(arr);
        free(temp);
        return 1;
    }

    for (int i = 0; i < n; i++)
    {
        if (fscanf(input_file, "%d, ", &arr[i]) != 1)
        {
            printf("Error reading from file\n");
            fclose(input_file);
            free(arr);
            free(temp);
            return 1;
        }
    }

    hipMalloc(&d_arr, n * sizeof(int));
    hipMalloc(&temp, n * sizeof(int));

    hipMemcpy(d_arr, arr, n * sizeof(int), hipMemcpyHostToDevice);

    // Determine grid and block dimensions based on the input size
    int num_threads = min(n, THREADS_PER_BLOCK);
    int num_blocks = (n + num_threads - 1) / num_threads;
    printf("%d", num_blocks);
    clock_t total_time = 0;
    int num_trials = 100;

    for (int t = 0; t < num_trials; t++)
    {
        hipMemcpy(d_arr, arr, n * sizeof(int), hipMemcpyHostToDevice);

        clock_t start = clock();

        mergeSort<<<num_blocks, num_threads>>>(d_arr, temp, n);

        hipDeviceSynchronize();

        clock_t end = clock();

        double elapsed_time = ((double)(end - start) / CLOCKS_PER_SEC) * 1000.0;
        total_time += elapsed_time;

        printf("Trial %d: Sorting time - %.2f milliseconds\n", t + 1, elapsed_time);
    }

    double average_time = (double)total_time / num_trials;
    printf("\nAverage sorting time over %d trials: %.2f milliseconds\n", num_trials, average_time);

    hipMemcpy(arr, d_arr, n * sizeof(int), hipMemcpyDeviceToHost);

    FILE *output_file = fopen("sorted.txt", "w");
    if (output_file == NULL)
    {
        printf("Error opening file: sorted.txt\n");
        hipFree(d_arr);
        hipFree(temp);
        free(arr);
        free(temp);
        return 1;
    }

    for (int i = 0; i < n; i++)
    {
        fprintf(output_file, "%d, ", arr[i]);
    }

    fclose(output_file);
    hipFree(d_arr);
    hipFree(temp);
    free(arr);
    free(temp);

    return 0;
}
